#include "iostream"
#include "hip/hip_runtime.h"


__device__ float div(int x, float f) {
    return float(x) / f;
}

template <typename OUT, typename... ARGS>
__global__ void sum(float(*func)(OUT, ARGS...), OUT* outputs, int size, float* dest, ARGS... args) {
    float s = 0.0;
    for (int i = 0 ; i < size ; ++i) {
        s += (*func)(outputs[i], args...);
    }
    *dest = s;
    //*dest = 0.0;
}

template <typename... Types>
__global__ void setup_kernel(float (**my_callback)(Types...)){
  *my_callback = &div;
}

void check_error() {
    hipDeviceSynchronize();

    hipError_t e = hipGetLastError();
    if (e != hipSuccess) {
        printf("Cuda failure: '%s'\n", hipGetErrorString(e));
        throw "";
    }
}

int main(void) {
    int vals[] = { 1, 2, 3 };
    int* device_vals;
    hipMalloc(&device_vals, 3 * sizeof(int));
    hipMemcpy(device_vals, vals, 3 * sizeof(int), hipMemcpyHostToDevice);

    float(*local_div)(int, float);
    float(**device_div)(int, float);
    hipMalloc(&device_div, sizeof(void *));

    setup_kernel<<<1, 1>>>((float(**)(int, float))device_div);
    hipMemcpy((void *)&local_div, (void *)device_div, sizeof(void *), hipMemcpyDeviceToHost);

    float *answer;
    hipMalloc(&answer, sizeof(float));

    sum<<<1, 1>>>(local_div, device_vals, 3, answer, (float) 2.0);
    check_error();

    float local_answer = 1.0;
    hipMemcpy(&local_answer, answer, sizeof(float), hipMemcpyDeviceToHost);

    std::cout << local_answer;
}
