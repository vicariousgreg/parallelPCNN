#include "iostream"
#include "hip/hip_runtime.h"


__device__ float divide(int x, float f) {
    return float(x) / f;
}

__device__ float (*divide_ptr)(int, float) = divide;

__device__ float mult(int x, float f) {
    return x * f;
}

__device__ float (*mult_ptr)(int, float) = mult;


template <typename OUT, typename... ARGS>
__global__ void sum(float(*func)(OUT, ARGS...), OUT* outputs, int size, float* dest, ARGS... args) {
    float s = 0.0;
    for (int i = 0 ; i < size ; ++i) {
        s += (*func)(outputs[i], args...);
    }
    *dest = s;
}

void check_error() {
    hipDeviceSynchronize();

    hipError_t e = hipGetLastError();
    if (e != hipSuccess) {
        printf("Cuda failure: '%s'\n", hipGetErrorString(e));
        throw "";
    }
}

int main(void) {
    // Set up testing data
    int vals[] = { 1, 2, 3 };
    int* device_vals;
    hipMalloc(&device_vals, 3 * sizeof(int));
    hipMemcpy(device_vals, vals, 3 * sizeof(int), hipMemcpyHostToDevice);

    // Set up space for the result
    float local_answer = 0.0;
    float *device_answer;
    hipMalloc(&device_answer, sizeof(float));

    // DIVIDE
    float(*local_div)(int, float);
    hipMemcpyFromSymbol(&local_div, HIP_SYMBOL(divide_ptr), sizeof(void *));
    sum<int, float><<<1, 1>>>(local_div, device_vals, 3, device_answer, (float) 2.0);
    hipMemcpy(&local_answer, device_answer, sizeof(float), hipMemcpyDeviceToHost);
    check_error();
    std::cout << local_answer << "\n";

    // MULT
    float(*local_mult)(int, float);
    hipMemcpyFromSymbol(&local_mult, HIP_SYMBOL(mult_ptr), sizeof(void *));
    sum<int, float><<<1, 1>>>(local_mult, device_vals, 3, device_answer, (float) 2.0);
    hipMemcpy(&local_answer, device_answer, sizeof(float), hipMemcpyDeviceToHost);
    check_error();
    std::cout << local_answer << "\n";
}
